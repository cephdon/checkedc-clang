
#include <hip/hip_runtime.h>
// Checked C extension is not supported for CUDA.   Make sure driver
// rejects the flag.
//
// RUN: not %clang -fcheckedc-extension -nocudalib -nocudainc %s 2>&1 | FileCheck %s
// CHECK: error: invalid argument '-fcheckedc-extension' not allowed with 'CUDA'
//
// Have clang compile this file as a C file.
// RUN: %clang -c -fcheckedc-extension -x c %s
//
// Have clang-cl compile this file as a C file.
// RUN: %clang_cl -c -Xclang -fcheckedc-extension /TC %s

extern void f() {}


